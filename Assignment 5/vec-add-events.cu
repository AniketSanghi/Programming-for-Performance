#include "hip/hip_runtime.h"
// Compile: nvcc -g -G -arch=sm_61 vec-add-events.cu -o vec-add-events

#include <cstddef>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using std::cout;
using std::endl;

const double THRESHOLD = 0.0000001;

__global__ void dkernel(float *a, float *b, float *c, size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    c[idx] = a[idx] + b[idx];
  }
}

void hkernel(float *a, float *b, float *c, size_t size) {
  for (int i = 0; i < size; i++) {
    c[i] = a[i] + b[i];
  }
}

__host__ void check_result(float *w_ref, float *w_opt, int N) {
  double maxdiff = 0.0, this_diff = 0.0;
  int numdiffs = 0;

  for (int i = 0; i < N; i++) {
    this_diff = w_ref[i] - w_opt[i];
    if (fabs(this_diff) > THRESHOLD) {
      numdiffs++;
      if (this_diff > maxdiff)
        maxdiff = this_diff;
    }
  }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over threshold " << THRESHOLD
         << "; Max Diff = " << maxdiff << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vecAdd_wrapper(float *h_a, float *h_b, float *h_c, size_t num) {
  float *dev_a = 0;
  float *dev_b = 0;
  float *dev_c = 0;

  size_t size = num * sizeof(float);
  hipError_t cudaStatus;
  hipEvent_t start, stop;
  int threadsPerBlock = 512;
  int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

  // Set device to use
  cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice() failed");
    goto Error;
  }

  cudaStatus = hipMalloc(&dev_a, size);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed");
    goto Error;
  }
  cudaStatus = hipMalloc(&dev_b, size);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed");
    goto Error;
  }
  cudaStatus = hipMalloc(&dev_c, size);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc() failed");
    goto Error;
  }

  cudaStatus = hipMemcpy(dev_a, h_a, size, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed");
    goto Error;
  }
  cudaStatus = hipMemcpy(dev_b, h_b, size, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed");
    goto Error;
  }

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);

  dkernel<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, dev_c, num);

  // hipDeviceSynchronize waits for the cuda kernel to finish
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr,
            "hipDeviceSynchronize() returned error code %d in dkernel "
            "execution\n",
            cudaStatus);
    goto Error;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // gpu_time will contain vector addition time at gpu without copying overhead
  float gpu_time;
  hipEventElapsedTime(&gpu_time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("Kernel time (ms) without copy overhead: %lf\n", gpu_time);

  cudaStatus = hipMemcpy(h_c, dev_c, size, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy() failed!");
  }

Error:
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return cudaStatus;
}

int main() {
  const int N = (1 << 24);
  float gpu_time;
  float cpu_time;

  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_gpu = (float *)malloc(N * sizeof(float));
  float *h_cpu = (float *)malloc(N * sizeof(float));

  int value_1 = 1;
  int value_2 = 2;

  std::fill_n(h_a, N, value_1);
  std::fill_n(h_b, N, value_2);
  std::fill_n(h_gpu, N, 0);
  std::fill_n(h_cpu, N, 0);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // Add in parallel from helper function with cuda_helper cuda kernel
  hipError_t cudaStatus = vecAdd_wrapper(h_a, h_b, h_gpu, N);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "add with cuda failed!");
    return EXIT_FAILURE;
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  // gpu_time will contain vector addition time at gpu with copying overhead
  hipEventElapsedTime(&gpu_time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("Kernel time (ms) with copy overhead: %lf\n", gpu_time);
  cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipDeviceReset() failed!");
    return EXIT_FAILURE;
  }

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  hkernel(h_a, h_b, h_cpu, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&cpu_time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("CPU time (ms): %lf\n", cpu_time);

  check_result(h_gpu, h_cpu, N);

  free(h_a);
  free(h_b);
  free(h_cpu);
  free(h_gpu);

  return EXIT_SUCCESS;
}
